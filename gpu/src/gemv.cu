#include <gpu/gemv.hpp>

#include <src_common.hpp>

namespace blas
{

milliseconds
gemv(const std::execution::parallel_gpu &,
     size_t N, size_t M,
     float * result,
     float alpha, const float * a, const float * x,
     float beta, const float * y)
{
    hipStream_t stream;
    cudaAssert(hipStreamCreate(&stream));

    hipblasHandle_t handle;
    cublasAssert(hipblasCreate(&handle));

    float * aDevice;
    float * xDevice;
    float * yDevice;
    cudaAssert(hipMalloc(&aDevice, N * M * sizeof(float)));
    cudaAssert(hipMalloc(&xDevice, M * sizeof(float)));
    cudaAssert(hipMalloc(&yDevice, N * sizeof(float)));

    hipEvent_t startGpu;
    hipEvent_t stopGpu;
    cudaAssert(hipEventCreate(&startGpu));
    cudaAssert(hipEventCreate(&stopGpu));

    cublasAssert(hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_ALLOWED));
    cublasAssert(hipblasSetMathMode(handle, HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION));
    cublasAssert(hipblasSetStream(handle, stream));

    cublasAssert(hipblasSetMatrixAsync(N, M, sizeof(float), a, M, aDevice, M, stream));
    cublasAssert(hipblasSetVectorAsync(M, sizeof(float), x, 1, xDevice, 1, stream));
    cublasAssert(hipblasSetVectorAsync(N, sizeof(float), y, 1, yDevice, 1, stream));

    cublasAssert(hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, aDevice, M, xDevice, 1, &beta, yDevice, 1));
    cudaAssert(hipEventRecord(startGpu, stream));
    cublasAssert(hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, aDevice, M, xDevice, 1, &beta, yDevice, 1));
    cudaAssert(hipEventRecord(stopGpu, stream));

    cublasAssert(hipblasGetVectorAsync(N, sizeof(float), yDevice, 1, result, 1, stream));

    cudaAssert(hipStreamSynchronize(stream));

    float gpuMs;
    cudaAssert(hipEventElapsedTime(&gpuMs, startGpu, stopGpu));

    cudaAssert(hipDeviceSynchronize());

    cudaAssert(hipEventDestroy(stopGpu));
    cudaAssert(hipEventDestroy(startGpu));

    cudaAssert(hipFree(aDevice));
    cudaAssert(hipFree(xDevice));
    cudaAssert(hipFree(yDevice));

    cublasAssert(hipblasDestroy(handle));

    cudaAssert(hipStreamDestroy(stream));

    return milliseconds{gpuMs};
}

}