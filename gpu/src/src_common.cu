#include <src_common.hpp>

#include <cassert>

namespace blas
{

void
cudaAssert(hipError_t error)
{
    if (error != hipSuccess) {
        assert(error == hipSuccess);
    }
}

void
cublasAssert(hipblasStatus_t error)
{
    if (error != HIPBLAS_STATUS_SUCCESS) {
        assert(error == HIPBLAS_STATUS_SUCCESS);
    }
}

}