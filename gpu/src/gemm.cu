#include <gpu/gemm.hpp>

#include <src_common.hpp>

namespace blas
{

milliseconds
gemm(const std::execution::parallel_gpu &,
     size_t N, size_t M, size_t P,
     float * result,
     float alpha, const float * a, const float * b,
     float beta, const float * c)
{
    hipStream_t stream;
    cudaAssert(hipStreamCreate(&stream));

    hipblasLtHandle_t handle;
    cublasAssert(hipblasLtCreate(&handle));

    hipblasLtMatmulDesc_t operationDescription;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    cublasAssert(hipblasLtMatmulDescCreate(&operationDescription, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    cublasAssert(hipblasLtMatmulDescSetAttribute(operationDescription, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(trans)));
    cublasAssert(hipblasLtMatmulDescSetAttribute(operationDescription, HIPBLASLT_MATMUL_DESC_TRANSB, &trans, sizeof(trans)));
    cublasAssert(hipblasLtMatmulDescSetAttribute(operationDescription, CUBLASLT_MATMUL_DESC_TRANSC, &trans, sizeof(trans)));

    hipblasLtMatrixLayout_t matrixADescription;
    hipblasLtMatrixLayout_t matrixBDescription;
    hipblasLtMatrixLayout_t matrixCDescription;
    hipblasLtOrder_t order = HIPBLASLT_ORDER_ROW;
    cublasAssert(hipblasLtMatrixLayoutCreate(&matrixADescription, HIP_R_32F, N, M, M));
    cublasAssert(hipblasLtMatrixLayoutSetAttribute(matrixADescription, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));
    cublasAssert(hipblasLtMatrixLayoutCreate(&matrixBDescription, HIP_R_32F, M, P, P));
    cublasAssert(hipblasLtMatrixLayoutSetAttribute(matrixBDescription, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));
    cublasAssert(hipblasLtMatrixLayoutCreate(&matrixCDescription, HIP_R_32F, N, P, P));
    cublasAssert(hipblasLtMatrixLayoutSetAttribute(matrixCDescription, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order)));

    hipblasLtMatmulPreference_t preference;
    size_t workspaceSize = 1024 * 1024 * sizeof(float );
    cublasAssert(hipblasLtMatmulPreferenceCreate(&preference));
    cublasAssert(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    int returnedResults{};
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    cublasAssert(hipblasLtMatmulAlgoGetHeuristic(
        handle, operationDescription,
        matrixADescription, matrixBDescription, matrixCDescription, matrixCDescription,
        preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        cublasAssert(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    float * aDevice;
    float * bDevice;
    float * cDevice;
    cudaAssert(hipMalloc(&aDevice, N * M * sizeof(float)));
    cudaAssert(hipMalloc(&bDevice, M * P * sizeof(float)));
    cudaAssert(hipMalloc(&cDevice, N * P * sizeof(float)));

    float * workspace;
    cudaAssert(hipMalloc(&workspace, workspaceSize));

    cudaAssert(hipMemcpyAsync(aDevice, a, N * M * sizeof(float), hipMemcpyHostToDevice, stream));
    cudaAssert(hipMemcpyAsync(bDevice, b, M * P * sizeof(float), hipMemcpyHostToDevice, stream));
    cudaAssert(hipMemcpyAsync(cDevice, c, N * P * sizeof(float), hipMemcpyHostToDevice, stream));

    hipEvent_t startGpu;
    hipEvent_t stopGpu;
    cudaAssert(hipEventCreate(&startGpu));
    cudaAssert(hipEventCreate(&stopGpu));

    cudaAssert(hipEventRecord(startGpu, stream));
    cublasAssert(hipblasLtMatmul(handle,
                                operationDescription,
                                &alpha,
                                aDevice,
                                matrixADescription,
                                bDevice,
                                matrixBDescription,
                                &beta,
                                cDevice,
                                matrixCDescription,
                                cDevice,
                                matrixCDescription,
                                &heuristicResult.algo,
                                workspace,
                                workspaceSize,
                                stream));
    cudaAssert(hipEventRecord(stopGpu, stream));

    cudaAssert(hipMemcpyAsync(result, cDevice, N * P * sizeof(float), hipMemcpyDeviceToHost, stream));

    cudaAssert(hipStreamSynchronize(stream));

    float gpuMs;
    cudaAssert(hipEventElapsedTime(&gpuMs, startGpu, stopGpu));

    cudaAssert(hipDeviceSynchronize());

    cudaAssert(hipEventDestroy(stopGpu));
    cudaAssert(hipEventDestroy(startGpu));

    cudaAssert(hipFree(workspace));

    cudaAssert(hipFree(aDevice));
    cudaAssert(hipFree(bDevice));
    cudaAssert(hipFree(cDevice));


    cublasAssert(hipblasLtMatmulPreferenceDestroy(preference));
    cublasAssert(hipblasLtMatrixLayoutDestroy(matrixCDescription));
    cublasAssert(hipblasLtMatrixLayoutDestroy(matrixBDescription));
    cublasAssert(hipblasLtMatrixLayoutDestroy(matrixADescription));
    cublasAssert(hipblasLtMatmulDescDestroy(operationDescription));
    cublasAssert(hipblasLtDestroy(handle));

    cudaAssert(hipStreamDestroy(stream));

    return milliseconds{gpuMs};

    /*
    cublasAssert(hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_ALLOWED));
    cublasAssert(hipblasSetMathMode(handle, HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION));
    cublasAssert(hipblasSetStream(handle, stream));
    cublasAssert(order(handle, stream));

    float * aDevice;
    float * bDevice;
    float * cDevice;
    cudaAssert(hipMalloc(&aDevice, N * M * sizeof(float)));
    cudaAssert(hipMalloc(&bDevice, M * P * sizeof(float)));
    cudaAssert(hipMalloc(&cDevice, N * P * sizeof(float)));

    float * aHost;
    float * bHost;
    float * cHost;
    cudaAssert(hipHostMalloc(&aHost, N * M * sizeof(float)));
    cudaAssert(hipHostMalloc(&bHost, M * P * sizeof(float)));
    cudaAssert(hipHostMalloc(&cHost, N * P * sizeof(float)));



    cublasAssert(hipblasSetMatrixAsync(N, M, sizeof(float), a, M, aDevice, M, stream));
    cublasAssert(hipblasSetMatrixAsync(M, P, sizeof(float), b, P, bDevice, P, stream));
    cublasAssert(hipblasSetMatrixAsync(N, P, sizeof(float), c, P, cDevice, P, stream));

    hipEvent_t startGpu;
    hipEvent_t stopGpu;
    cudaAssert(hipEventCreate(&startGpu));
    cudaAssert(hipEventCreate(&stopGpu));

    cublasAssert(
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, P, M, &alpha, aDevice, HIP_R_32F, M, bDevice,
                     HIP_R_32F, P,
                     &beta, cDevice, HIP_R_32F, P, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT
        ));
    cudaAssert(hipEventRecord(startGpu, stream));
    cublasAssert(
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, P, M, &alpha, aDevice, HIP_R_32F, M, bDevice,
                     HIP_R_32F, P,
                     &beta, cDevice, HIP_R_32F, P, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT
        ));
    cudaAssert(hipEventRecord(stopGpu, stream));

    cublasAssert(hipblasGetMatrixAsync(N, P, sizeof(float), cDevice, P, result, P, stream));

    cudaAssert(hipStreamSynchronize(stream));

    float gpuMs;
    cudaAssert(hipEventElapsedTime(&gpuMs, startGpu, stopGpu));

    cudaAssert(hipDeviceSynchronize());

    cudaAssert(hipEventDestroy(stopGpu));
    cudaAssert(hipEventDestroy(startGpu));

    cudaAssert(hipFree(aDevice));
    cudaAssert(hipFree(bDevice));
    cudaAssert(hipFree(cDevice));

    cublasAssert(hipblasDestroy(handle));

    cudaAssert(hipStreamDestroy(stream));

     return milliseconds{gpuMs};
*/
}

}
